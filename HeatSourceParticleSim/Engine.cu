#include "hip/hip_runtime.h"
#include "Engine.cuh"

Engine::Engine()
{
	Particle p[ParticleCount];

	hipMalloc(&d_CellParticleCount, sizeof(int) * GridWidth * GridHeight);
	hipMalloc(&d_CellParticleIDs, sizeof(int) * GridWidth * GridHeight * MaxParticlesPerCell);
	hipMalloc(&d_Particles, sizeof(Particle) * ParticleCount);
	hipMemcpy(d_Particles, p, sizeof(Particle) * ParticleCount, hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	wm = new WindowManager();

	TickClock.restart();
	dt = 1;
}

void Engine::Tick()
{
	dt = TickClock.restart().asSeconds();
	if (dt >= 0.01) return;
	//ResetGrid << <(GridCount + 1023) / 1024, 1024 >> > (d_CellParticleCount);
	//SetupGrid << <(ParticleCount + 1023) / 1024, 1024 >> > (d_CellParticleCount, d_CellParticleIDs, d_Particles);
	ProcessParticle << < (ParticleCount + 1023) / 1024, 1024 >> > (d_Particles, dt);
	ERRORCHECKLAST;
	hipDeviceSynchronize();
	ProcessParticleVelocity << < (ParticleCount + 1023) / 1024, 1024 >> > (d_Particles, dt);
	ERRORCHECKLAST;
	DrawParticle << < (ParticleCount + 1023) / 1024, 1024 >> > (d_Particles, wm->d_pixels);
	ERRORCHECKLAST;
}

__global__ void ResetGrid(int* GridStart)
{
	GETID(GridCount);
	GridStart[id] = 0;
}

__global__ void SetupGrid(int* g, int* gi, Particle* p)
{
	GETID(ParticleCount);
	Particle* pa = &p[id];
	int x = pa->Location.x / WindowWidth * GridWidth;
	int y = pa->Location.y / WindowHeight * GridHeight;
	int c = GridWidth * y + x;
	pa->Cell = c;
	if (g[c] >= MaxParticlesPerCell) return;
	gi[c * MaxParticlesPerCell + g[c]] = id;
	++g[c];
}